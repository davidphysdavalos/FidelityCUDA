#include "hip/hip_runtime.h"
#include <iostream>
#include <cpp/dev_random.cpp>
#include <tclap/CmdLine.h>
#include <itpp/itbase.h>
#include <itpp/stat/histogram.h>
#include "cpp/RMT.cpp"	
#include <cpp/itpp_ext_math.cpp>
#include <cpp/spinchain.cpp>
#include <itpp/stat/misc_stat.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include "cuda_functions.cu"
#include "cuda_utils.cu"
#include "ev_routines.cu"
#include "cfp_routines.cu"

//using namespace std; 
//using namespace itpp;
//using namespace itppextmath;
//using namespace cfpmath;
//using namespace spinchain;


TCLAP::CmdLine cmd("Command description message", ' ', "0.1");
TCLAP::ValueArg<string> optionArg("o","option", "Option" ,false,"normalito", "string",cmd);
TCLAP::ValueArg<string> optionArg2("","option2", "Option2" ,false,"fidelity", "string",cmd);
TCLAP::ValueArg<unsigned int> seed("s","seed", "Random seed [0 for urandom]",false, 243243,"unsigned int",cmd);
TCLAP::ValueArg<int> qubits("q","qubits", "number of qubits",false, 4,"int",cmd);
TCLAP::ValueArg<double> J("J","ising_coupling", "Ising interaction in the z-direction",false, 1.0,"double",cmd);
TCLAP::ValueArg<double> bx("","bx", "Magnetic field in x direction",false, 1.4,"double",cmd);
TCLAP::ValueArg<double> by("","by", "Magnetic field in y direction",false, 0.,"double",cmd);
TCLAP::ValueArg<double> bz("","bz", "Magnetic field in z direction",false, 1.4,"double",cmd);
TCLAP::ValueArg<double> theta("","theta", "polar angle",false, 1.0,"double",cmd);
TCLAP::ValueArg<double> phi("","phi", "azimultal angle",false, 1.0,"double",cmd);
TCLAP::ValueArg<double> deltabx("","deltabx", "perturbation",false, 0.1,"double",cmd);
TCLAP::ValueArg<int> steps("","steps","steps",false, 100,"int",cmd);
TCLAP::ValueArg<double> Jpert("","Jpert","Perturbation on Ising",false, 0.0,"double",cmd);
TCLAP::ValueArg<int> dev("","dev", "Gpu to be used, 0 for c20, 1 para la jodida",false, 0,"int",cmd);


int main(int argc, char* argv[])
{

cmd.parse( argc, argv );
cout.precision(17);
hipSetDevice(dev.getValue());

// {{{ Set seed for random
unsigned int semilla=seed.getValue();
if (semilla == 0){
  Random semilla_uran; semilla=semilla_uran.strong();
} 
itpp::RNG_reset(semilla);
// }}}

itpp::vec b(3), bpert(3), bzeros(3); 
b(0)=bx.getValue(); 
b(1)=by.getValue();
b(2)=bz.getValue();
bzeros=b-b;
bpert=b;
bpert(0)=b(0)+deltabx.getValue();
string option=optionArg.getValue();
string option2=optionArg2.getValue();

itpp::cvec state, staterev, qustate;

//ofstream fidelity;
//fidelity.open("fidelity.dat");

//qustate=RandomState(64);

//int dim=pow_2(qubits.getValue());

qustate=itppextmath::BlochToQubit(theta.getValue(),phi.getValue());

//qustate=RandomState(2);

//for(int i=0; i<qubits.getValue()+1;i++){

//list(i)=qustate;

//}

if(option=="normalito")
	state=itppextmath::TensorPow(qustate,qubits.getValue());
	
if(option=="randU")
	state=RMT::RandomCUE(pow(2, qubits.getValue()))*itppextmath::TensorPow(qustate,qubits.getValue());
	
if(option=="klimov")
	state=itppextmath::TensorProduct(itppextmath::TensorProduct(itppextmath::TensorPow(qustate,3),itppextmath::sigma(1)*qustate),itppextmath::TensorPow(qustate,qubits.getValue()-4));
	
if(option=="klimovy")
	state=itppextmath::TensorProduct(itppextmath::TensorProduct(itppextmath::TensorPow(qustate,3),itppextmath::sigma(2)*qustate),itppextmath::TensorPow(qustate,qubits.getValue()-4));
	
if(option=="klimov2")
		state=itppextmath::TensorProduct(itppextmath::TensorProduct(itppextmath::TensorPow(qustate,2),itppextmath::TensorPow(itppextmath::sigma(1)*qustate,2)),itppextmath::TensorPow(qustate,qubits.getValue()-4));


//cout<< qustate ;

staterev=state;

double Jrev=J.getValue()+Jpert.getValue();

if(option2=="fidelity"){

itpp::vec list(steps.getValue());

for(int i=0;i<steps.getValue();i++){

list(i)=pow( abs( dot( conj(staterev),state)),2);

//cout<< pow( abs( dot( conj(staterev),state)),2) <<endl;

std::cout << list(i) <<endl;
// cout<< i<< " " << list(i) <<endl;

list(i)=sqrt(list(i));

itppcuda::apply_floquet(state, J.getValue(), b);

itppcuda::apply_floquet(staterev, Jrev, bpert); 

//cout<<abs(dot(conj(staterev),state))<<endl;

//fidelity<<pow(abs(dot(conj(staterev),state)),2)<<endl;

}
 
//fidelity.close();

//cout << staterev;

std::cout<< itppextmath::sum_positive_derivatives(list)<< endl;
}

if(option2=="correlacion"){
	
itpp::cvec list(steps.getValue());

itpp::cvec init=state;

for(int i=0;i<steps.getValue();i++){

list(i)=dot(conj(init),state);

std::cout << real(list(i)) << " " << imag(list(i)) <<endl;

//cout << list <<endl;

itppcuda::apply_floquet(state, J.getValue(), b);
}
}

if(option2=="fidelityandipr"){

itpp::vec listfidel(steps.getValue());

itpp::cvec listcorr(steps.getValue());

itpp::cvec init=state;

for(int i=0;i<steps.getValue();i++){

listfidel(i)=pow( abs( dot( conj(staterev),state)),2);

listcorr(i)=pow(abs(dot(conj(init),state)),2);

//cout<< pow( abs( dot( conj(staterev),state)),2) <<endl;

std::cout << listfidel(i) <<endl;
// cout<< i<< " " << list(i) <<endl;

listfidel(i)=sqrt(listfidel(i));

itppcuda::apply_floquet(state, J.getValue(), b);

itppcuda::apply_floquet(staterev, J.getValue(), b);

itppcuda::apply_floquet(staterev, Jpert.getValue(), bzeros);

//cout<<abs(dot(conj(staterev),state))<<endl;

//fidelity<<pow(abs(dot(conj(staterev),state)),2)<<endl;

}
 
//fidelity.close();

//cout << staterev;

cout<< itppextmath::sum_positive_derivatives(listfidel)<< endl;

cout<< real(mean(listcorr))<< endl;
}


}
