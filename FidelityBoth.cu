#include "hip/hip_runtime.h"
#include <iostream>
#include <cpp/dev_random.cpp>
#include <tclap/CmdLine.h>
#include <itpp/itbase.h>
#include <itpp/stat/histogram.h>
#include "cpp/RMT.cpp"	
#include <cpp/itpp_ext_math.cpp>
#include <cpp/spinchain.cpp>
#include <itpp/stat/misc_stat.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include "cuda_functions.cu"
#include "cuda_utils.cu"
#include "ev_routines.cu"
#include "cfp_routines.cu"

/*
using namespace std; 
using namespace itpp;
using namespace itppextmath;
using namespace cfpmath;
using namespace spinchain;
using namespace RMT;
/**/

TCLAP::CmdLine cmd("Command description message", ' ', "0.1");
TCLAP::ValueArg<string> optionArg("o","option", "Option" ,false,"normalito", "string",cmd);
TCLAP::ValueArg<string> optionArg2("","option2", "Option2" ,false,"fidelity", "string",cmd);
TCLAP::ValueArg<unsigned int> seed("s","seed", "Random seed [0 for urandom]",false, 243243,"unsigned int",cmd);
TCLAP::ValueArg<int> qubits("q","qubits", "number of qubits",false, 4,"int",cmd);
TCLAP::ValueArg<double> J("J","ising_coupling", "Ising interaction in the z-direction",false, 1.0,"double",cmd);
TCLAP::ValueArg<double> bx("","bx", "Magnetic field in x direction",false, 1.4,"double",cmd);
TCLAP::ValueArg<double> by("","by", "Magnetic field in y direction",false, 0.,"double",cmd);
TCLAP::ValueArg<double> bz("","bz", "Magnetic field in z direction",false, 1.4,"double",cmd);
TCLAP::ValueArg<double> theta("","theta", "polar angle",false, 1.0,"double",cmd);
TCLAP::ValueArg<double> phi("","phi", "azimultal angle",false, 1.0,"double",cmd);
TCLAP::ValueArg<double> deltabx("","deltabx", "perturbation",false, 0.1,"double",cmd);
TCLAP::ValueArg<int> steps("","steps","steps",false, 100,"int",cmd);
TCLAP::ValueArg<double> Jpert("","Jpert","Perturbation on Ising",false, 0.0,"double",cmd);
TCLAP::ValueArg<double> Jinhompert("","Jinhompert","Inhomogeneous perturbation on Ising on 0-1 interaction",false, 0.0,"double",cmd);
TCLAP::ValueArg<double> deltabxinhom("","deltabxinhom", "perturbation al campo solo en el qubit 0",false, 0.0,"double",cmd);


int main(int argc, char* argv[])
{

cmd.parse( argc, argv );
cout.precision(12);
hipSetDevice(dev.getValue());

// {{{ Set seed for random
unsigned int semilla=seed.getValue();
if (semilla == 0){
  Random semilla_uran; semilla=semilla_uran.strong();
} 
itpp::RNG_reset(semilla);
// }}}

itpp::vec b(3), bpert(3), bpertrev(3), binhom(3), binhomrev(3);
b(0)=bx.getValue(); 
b(1)=by.getValue();
b(2)=bz.getValue();
bpert=b;
bpertrev=b;
bpert(0)=b(0)+deltabx.getValue();
bpertrev(0)=b(0)-deltabx.getValue();
binhom=bpert;
binhomrev=bpertrev;
binhom(0)=bpert(0)+deltabxinhom.getValue();
binhomrev(0)=bpertrev(0)-deltabxinhom.getValue();
string option=optionArg.getValue();
string option2=optionArg2.getValue();

itpp::cvec state, staterev, qustate;

qustate=itppextmath::BlochToQubit(theta.getValue(),phi.getValue());

//qustate=RandomState(64);

//int dim=pow_2(qubits.getValue());

qustate=BlochToQubit(theta.getValue(),phi.getValue());

//qustate=RandomState(2);

//for(int i=0; i<qubits.getValue()+1;i++){

//list(i)=qustate;

//}

if(option=="normalito")
	state=itppextmath::TensorPow(qustate,qubits.getValue());
	
if(option=="randU")
	state=RMT::RandomCUE(pow(2, qubits.getValue()))*itppextmath::TensorPow(qustate,qubits.getValue());
	
if(option=="klimov")
	state=itppextmath::TensorProduct(itppextmath::TensorProduct(itppextmath::TensorPow(qustate,3),itppextmath::sigma(1)*qustate),itppextmath::TensorPow(qustate,qubits.getValue()-4));
	
if(option=="klimovy")
	state=itppextmath::TensorProduct(itppextmath::TensorProduct(itppextmath::TensorPow(qustate,3),itppextmath::sigma(2)*qustate),itppextmath::TensorPow(qustate,qubits.getValue()-4));
	
if(option=="klimov2")
		state=itppextmath::TensorProduct(itppextmath::TensorProduct(itppextmath::TensorPow(qustate,2),itppextmath::TensorPow(itppextmath::sigma(1)*qustate,2)),itppextmath::TensorPow(qustate,qubits.getValue()-4));
	
if(option=="random")
	state=itppextmath::RandomState(pow(2,qubits.getValue()));

//cout<< qustate ;

staterev=state;

double Jrev=J.getValue()+Jpert.getValue();


if(option2=="fidelity"){

itpp::vec list(steps.getValue());

for(int i=0;i<steps.getValue();i++){

list(i)=pow( abs( dot( conj(staterev),state)),2);

//cout<< pow( abs( dot( conj(staterev),state)),2) <<endl;

cout << list(i) <<endl;
// cout<< i<< " " << list(i) <<endl;

list(i)=sqrt(list(i));

apply_ising_chain_inhom(state, J.getValue()+Jpert.getValue(), J.getValue()+Jinhompert.getValue()+Jpert.getValue());

apply_inhomogeneous_kick(state, bpert, binhom);

apply_ising_chain_inhom(staterev, J.getValue()-Jpert.getValue(), J.getValue()-Jinhompert.getValue()-Jpert.getValue());

apply_inhomogeneous_kick(staterev, bpertrev, binhomrev);

}
 
//fidelity.close();

//cout << staterev;

cout<< itppextmath::sum_positive_derivatives(list)<< endl;
}
//cout<<state<<endl;
if(option2=="correlacion"){
	
cvec list(steps.getValue());

cvec init=state;

for(int i=0;i<steps.getValue();i++){

list(i)=dot(conj(init),state);

cout << real(list(i)) << " " << imag(list(i)) <<endl;

//cout << list <<endl;

apply_ising_inhom(state, J.getValue(), J.getValue()+Jinhompert.getValue());

apply_magnetic_inhom(state, bpert, binhom);
}
}
if(option2=="fidelityandipr"){

vec listfidel(steps.getValue());

cvec listcorr(steps.getValue());

cvec init=state;

for(int i=0;i<steps.getValue();i++){

listfidel(i)=pow( abs( dot( conj(staterev),state)),2);

listcorr(i)=pow(abs(dot(conj(init),state)),2);

//cout<< pow( abs( dot( conj(staterev),state)),2) <<endl;

cout << listfidel(i) <<endl;
// cout<< i<< " " << list(i) <<endl;

listfidel(i)=sqrt(listfidel(i));

apply_ising_inhom(state, J.getValue()+Jpert.getValue(), J.getValue()+Jinhompert.getValue()+Jpert.getValue());

apply_magnetic_inhom(state, bpert, binhom);

apply_ising_inhom(staterev, J.getValue()-Jpert.getValue(), J.getValue()-Jinhompert.getValue()-Jpert.getValue());

apply_magnetic_inhom(staterev, bpertrev, binhomrev);

//cout<<abs(dot(conj(staterev),state))<<endl;

//fidelity<<pow(abs(dot(conj(staterev),state)),2)<<endl;

}
 
//fidelity.close();

//cout << staterev;

cout<< sum_positive_derivatives(listfidel)<< endl;

cout<< real(mean(listcorr))<< endl;
}

//cout<<binhom<<" "<<binhomrev<<endl;

}
