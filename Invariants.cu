#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <itpp/itbase.h>
#include <itpp_ext_math.cpp>
#include <math.h>
#include <tclap/CmdLine.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include "tools.cpp"
#include <spinchain.cpp>
#include "cuda_functions.cu"
#include "cuda_utils.cu"
#include "ev_routines.cu"
#include "cfp_routines.cu"
#include <tclap/CmdLine.h>




  TCLAP::CmdLine cmd("Command description message", ' ', "0.1");
  TCLAP::ValueArg<int> qubits("q","qubits", "Number of qubits",false, 3,"int",cmd);
  TCLAP::ValueArg<double> ising("","ising_z", "Ising interaction in the z-direction",false, 1,"double",cmd);
  TCLAP::ValueArg<double> k("","k", "qusimomentum number",false,0,"double",cmd);
  TCLAP::ValueArg<int> dev("","dev", "Gpu to be used, 0 for k20, 1 for c20",false, 0,"int",cmd);
  TCLAP::ValueArg<int> symx("","symx", "If simetry on sigma_x is to be used ",false, 0,"int",cmd);	
  
    int main(int argc,char* argv[]) {
		
      cout.precision(17);
      hipSetDevice(dev.getValue());
      itpp::RNG_randomize();
      cmd.parse(argc,argv);
      
      itpp::cmat vec=evcuda::invariant_vectors(qubits.getValue(),qubits.getValue(), k.getValue(), 0, symx.getValue());
      
      //int dim=itpp::rank(vec);
      cout<<vec<<endl;
      //cout<< dim<<" perame "<<endl;
      //cout<< vec<<" perame "<<endl;
      //cout<< vec(1,1)<<" perame otra vez"<<endl;
     // for (int i=0; i<vec.rows(); i++){
      //for (int j=0; j<vec.cols(); j++)
	//			cout<< itppextmath::Chop(real(vec(i,j))) <<" "<< itppextmath::Chop(imag(vec(i,j)))<<endl;
//	}
   //pow(qubits.getValue(),2)   
  }
