#include "hip/hip_runtime.h"
#include <iostream>
#include <cpp/dev_random.cpp>
#include <tclap/CmdLine.h>
#include <itpp/itbase.h>
#include <itpp/stat/histogram.h>
#include "cpp/RMT.cpp"	
#include <cpp/itpp_ext_math.cpp>
#include <cpp/spinchain.cpp>
#include <itpp/stat/misc_stat.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include "cuda_functions.cu"
#include "cuda_utils.cu"
#include "ev_routines.cu"
#include "cfp_routines.cu"

//using namespace std; 
//using namespace itpp;
//using namespace itppextmath;
//using namespace cfpmath;
//using namespace spinchain;


TCLAP::CmdLine cmd("Command description message", ' ', "0.1");
TCLAP::ValueArg<string> optionArg("o","option", "Option" ,false,"normalito", "string",cmd);
TCLAP::ValueArg<unsigned int> seed("s","seed", "Random seed [0 for urandom]",false, 243243,"unsigned int",cmd);
TCLAP::ValueArg<int> qubits("q","qubits", "number of qubits",false, 4,"int",cmd);
TCLAP::ValueArg<double> J("J","ising_coupling", "Ising interaction in the z-direction",false, 1.0,"double",cmd);
TCLAP::ValueArg<double> bx("","bx", "Magnetic field in x direction",false, 1.4,"double",cmd);
TCLAP::ValueArg<double> by("","by", "Magnetic field in y direction",false, 0.,"double",cmd);
TCLAP::ValueArg<double> bz("","bz", "Magnetic field in z direction",false, 1.4,"double",cmd);
TCLAP::ValueArg<double> theta("","theta", "polar angle",false, 1.0,"double",cmd);
TCLAP::ValueArg<double> phi("","phi", "azimultal angle",false, 1.0,"double",cmd);
TCLAP::ValueArg<double> deltabx("","deltabx", "perturbation",false, 0.1,"double",cmd);
TCLAP::ValueArg<int> steps("","steps","steps",false, 100,"int",cmd);
TCLAP::ValueArg<double> Jpert("","Jpert","Perturbation on Ising",false, 0.0,"double",cmd);
TCLAP::ValueArg<int> dev("","dev", "Gpu to be used, 0 for k20, 1 for c20",false, 1,"int",cmd);


int main(int argc, char* argv[])
{

cmd.parse( argc, argv );
cout.precision(17);
hipSetDevice(dev.getValue());

// {{{ Set seed for random
unsigned int semilla=seed.getValue();
if (semilla == 0){
  Random semilla_uran; semilla=semilla_uran.strong();
} 
itpp::RNG_reset(semilla);
// }}}

itpp::vec b(3), bpert(3); 
b(0)=bx.getValue(); 
b(1)=by.getValue();
b(2)=bz.getValue();
bpert=b;
bpert(0)=b(0)+deltabx.getValue();
string option=optionArg.getValue();

itpp::cvec state, staterev, qustate;

//ofstream fidelity;
//fidelity.open("fidelity.dat");

//qustate=RandomState(64);

//int dim=pow_2(qubits.getValue());

qustate=itppextmath::BlochToQubit(theta.getValue(),phi.getValue());

//qustate=RandomState(2);

//for(int i=0; i<qubits.getValue()+1;i++){

//list(i)=qustate;

//}

if(option=="normalito")
	state=itppextmath::TensorPow(qustate,qubits.getValue());
	
if(option=="randU")
	state=RMT::RandomCUE(pow(2, qubits.getValue()))*itppextmath::TensorPow(qustate,qubits.getValue());
	
if(option=="klimov")
	state=itppextmath::TensorProduct(itppextmath::TensorProduct(itppextmath::TensorPow(qustate,3),itppextmath::sigma(1)*qustate),itppextmath::TensorPow(qustate,qubits.getValue()-4));
	
if(option=="klimovy")
	state=itppextmath::TensorProduct(itppextmath::TensorProduct(itppextmath::TensorPow(qustate,3),itppextmath::sigma(2)*qustate),itppextmath::TensorPow(qustate,qubits.getValue()-4));
	
if(option=="klimov2")
		state=itppextmath::TensorProduct(itppextmath::TensorProduct(itppextmath::TensorPow(qustate,2),itppextmath::TensorPow(itppextmath::sigma(1)*qustate,2)),itppextmath::TensorPow(qustate,qubits.getValue()-4));


//cout<< qustate ;

staterev=state;

double Jrev=J.getValue()+Jpert.getValue();

itpp::vec list(steps.getValue());

for(int i=0;i<steps.getValue();i++){

list(i)=pow( abs( dot( conj(staterev),state)),2);

//cout<< pow( abs( dot( conj(staterev),state)),2) <<endl;

cout << list(i) <<endl;
// cout<< i<< " " << list(i) <<endl;

list(i)=sqrt(list(i));

itppcuda::apply_floquet(state, J.getValue(), b);

itppcuda::apply_floquet(staterev, Jrev, bpert); 

//cout<<abs(dot(conj(staterev),state))<<endl;

//fidelity<<pow(abs(dot(conj(staterev),state)),2)<<endl;

}
 
//fidelity.close();

//cout << staterev;

std::cout<< itppextmath::sum_positive_derivatives(list)<< endl;

//std::cout<<state<<endl;


}
