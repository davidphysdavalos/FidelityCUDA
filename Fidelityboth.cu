#include "hip/hip_runtime.h"
#include <iostream>
#include <cpp/dev_random.cpp>
#include <tclap/CmdLine.h>
#include <itpp/itbase.h>
#include <itpp/stat/histogram.h>
#include "cpp/RMT.cpp"	
#include <cpp/itpp_ext_math.cpp>
#include <cpp/spinchain.cpp>
#include <itpp/stat/misc_stat.h>
#include <fstream>
#include <hip/hip_runtime.h>
//#include "tools.cpp"
#include "cuda_functions.cu"
#include "cuda_utils.cu"
#include "ev_routines.cu"
#include "cfp_routines.cu"

//~ using namespace std; 
//~ using namespace itpp;
//~ using namespace itppextmath;
//~ using namespace cfpmath;
//~ using namespace spinchain;
//~ using namespace RMT;
		
	void apply_ising_chain_inhom(itpp::cvec& state,double ising, double Jinhom) { // {{{
		double* dev_R;
		double* dev_I;
		int l=state.size();
		int nqubits=log(l)/log(2);
		// cout << nqubits;
		int numthreads;
		int numblocks;
		choosenumblocks(l,numthreads,numblocks);
		//set_parameters(ising,b,icos,isin,kcos,ksin,bx,by,bz);

		itppcuda::itpp2cuda(state,&dev_R,&dev_I);


			for(int i=1;i<nqubits;i++) {
				Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%nqubits,dev_R,dev_I,cos(ising),sin(ising),l);
				//       cudaCheckError("ising",i);
			}
			Ui_kernel<<<numblocks,numthreads>>>(0,1,dev_R,dev_I,cos(Jinhom),sin(Jinhom),l);
		//cout<<dev_R<<" "<<dev_I<<endl;

		itppcuda::cuda2itpp(state,dev_R,dev_I);
		hipFree(dev_R);
		hipFree(dev_I);

	} // }}}
	
		void apply_inhomogeneous_kick(itpp::cvec& state ,itpp::vec b, itpp::vec binhom ){ // {{{
		double* dev_R;
		double* dev_I;
		int l=state.size();
		int nqubits=log(l)/log(2);
		// cout << nqubits;
		int numthreads;
		int numblocks;
		double theta=itpp::norm(b);
		double theta2=itpp::norm(binhom);
		choosenumblocks(l,numthreads,numblocks);

		itppcuda::itpp2cuda(state,&dev_R,&dev_I);
		
		Uk_kernel<<<numblocks,numthreads>>>(0,dev_R,dev_I,binhom(0)/theta2,binhom(1)/theta2,binhom(2)/theta2,cos(theta2),sin(theta2),l);

			for(int i=1;i<nqubits;i++) {
				Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,b(0)/theta,b(1)/theta,b(2)/theta,cos(theta),sin(theta),l);
				//       cudaCheckError("kick",i);
			}
		//cout<<dev_R<<" "<<dev_I<<endl;

		itppcuda::cuda2itpp(state,dev_R,dev_I);
		hipFree(dev_R);
		hipFree(dev_I);

	} 


TCLAP::CmdLine cmd("Command description message", ' ', "0.1");
TCLAP::ValueArg<string> optionArg("o","option", "Option" ,false,"normalito", "string",cmd);
TCLAP::ValueArg<string> optionArg2("","option2", "Option2" ,false,"fidelity", "string",cmd);
TCLAP::ValueArg<unsigned int> seed("s","seed", "Random seed [0 for urandom]",false, 243243,"unsigned int",cmd);
TCLAP::ValueArg<int> qubits("q","qubits", "number of qubits",false, 4,"int",cmd);
TCLAP::ValueArg<double> J("J","ising_coupling", "Ising interaction in the z-direction",false, 1.0,"double",cmd);
TCLAP::ValueArg<double> bx("","bx", "Magnetic field in x direction",false, 1.4,"double",cmd);
TCLAP::ValueArg<double> by("","by", "Magnetic field in y direction",false, 0.,"double",cmd);
TCLAP::ValueArg<double> bz("","bz", "Magnetic field in z direction",false, 1.4,"double",cmd);
TCLAP::ValueArg<double> theta("","theta", "polar angle",false, 1.0,"double",cmd);
TCLAP::ValueArg<double> phi("","phi", "azimultal angle",false, 1.0,"double",cmd);
TCLAP::ValueArg<double> deltabx("","deltabx", "perturbation campo en x",false, 0.0,"double",cmd);
TCLAP::ValueArg<double> deltabz("","deltabz", "perturbation campo en z",false, 0.0,"double",cmd);
TCLAP::ValueArg<int> steps("","steps","steps",false, 100,"int",cmd);
TCLAP::ValueArg<double> Jpert("","Jpert","Perturbation on Ising",false, 0.0,"double",cmd);
TCLAP::ValueArg<double> Jinhompert("","Jinhompert","Inhomogeneous perturbation on Ising on 0-1 interaction",false, 0.0,"double",cmd);
TCLAP::ValueArg<double> deltabxinhom("","deltabxinhom", "perturbation al campo solo en el qubit 0",false, 0.0,"double",cmd);
TCLAP::ValueArg<int> dev("","dev", "Gpu to be used, 0 for c20, 1 para la jodida",false, 0,"int",cmd);


int main(int argc, char* argv[])
{
cmd.parse( argc, argv );
cout.precision(12);
hipSetDevice(dev.getValue());

// {{{ Set seed for random
unsigned int semilla=seed.getValue();
if (semilla == 0){
  Random semilla_uran; semilla=semilla_uran.strong();
} 
itpp::RNG_reset(semilla);
// }}}

itpp::vec b(3), bpert(3), bpertrev(3), binhom(3), binhomrev(3);
b(0)=bx.getValue(); 
b(1)=by.getValue();
b(2)=bz.getValue();
bpert=b;
bpertrev=b;


bpert(0)=b(0)+deltabx.getValue();
// Para perturbacion en z
bpert(2)=b(2)+deltabz.getValue();


bpertrev(0)=b(0)-deltabx.getValue();
// Para perturbacion en z
bpertrev(2)=b(2)-deltabz.getValue();


binhom=bpert;
binhomrev=bpertrev;
binhom(0)=bpert(0)+deltabxinhom.getValue();
binhomrev(0)=bpertrev(0)-deltabxinhom.getValue();
string option=optionArg.getValue();
string option2=optionArg2.getValue();

itpp::cvec state, staterev, qustate;

qustate=itppextmath::BlochToQubit(theta.getValue(),phi.getValue());

{{{
if(option=="normalito")
	state=itppextmath::TensorPow(qustate,qubits.getValue());
	
if(option=="randU")
	state=RMT::RandomCUE(pow(2, qubits.getValue()))*itppextmath::TensorPow(qustate,qubits.getValue());
	
if(option=="klimov")
	state=itppextmath::TensorProduct(itppextmath::TensorProduct(itppextmath::TensorPow(qustate,3),itppextmath::sigma(1)*qustate),itppextmath::TensorPow(qustate,qubits.getValue()-4));
	
if(option=="klimovy")
	state=itppextmath::TensorProduct(itppextmath::TensorProduct(itppextmath::TensorPow(qustate,3),itppextmath::sigma(2)*qustate),itppextmath::TensorPow(qustate,qubits.getValue()-4));
	
if(option=="klimov2")
		state=itppextmath::TensorProduct(itppextmath::TensorProduct(itppextmath::TensorPow(qustate,2),itppextmath::TensorPow(itppextmath::sigma(1)*qustate,2)),itppextmath::TensorPow(qustate,qubits.getValue()-4));

if(option=="random")
	state=itppextmath::RandomState(pow(2,qubits.getValue()));

}}}
//cout<< qustate ;

staterev=state;

double Jrev=J.getValue()+Jpert.getValue();


if(option2=="fidelity"){

itpp::vec list(steps.getValue());

for(int i=0;i<steps.getValue();i++){

list(i)=pow( abs( dot( conj(staterev),state)),2);

//cout<< pow( abs( dot( conj(staterev),state)),2) <<endl;

cout << list(i) <<endl;
// cout<< i<< " " << list(i) <<endl;

list(i)=sqrt(list(i));

apply_ising_chain_inhom(state, J.getValue()+Jpert.getValue(), J.getValue()+Jinhompert.getValue()+Jpert.getValue());

apply_inhomogeneous_kick(state, bpert, binhom);

apply_ising_chain_inhom(staterev, J.getValue()-Jpert.getValue(), J.getValue()-Jinhompert.getValue()-Jpert.getValue());

apply_inhomogeneous_kick(staterev, bpertrev, binhomrev);

}
 
//fidelity.close();

//cout << staterev;

cout<< itppextmath::sum_positive_derivatives(list)<< endl;
}
if(option2=="fidelityandipr"){

itpp::vec listfidel(steps.getValue());

itpp::cvec listcorr(steps.getValue());

itpp::cvec init=state;

for(int i=0;i<steps.getValue();i++){

listfidel(i)=pow( abs( dot( conj(staterev),state)),2);

listcorr(i)=pow(abs(dot(conj(init),state)),2);

std::cout << listfidel(i) <<endl;

listfidel(i)=sqrt(listfidel(i));

apply_ising_chain_inhom(state, J.getValue()+Jpert.getValue(), J.getValue()+Jinhompert.getValue()+Jpert.getValue());

apply_inhomogeneous_kick(state, bpert, binhom);

apply_ising_chain_inhom(staterev, J.getValue()-Jpert.getValue(), J.getValue()-Jinhompert.getValue()-Jpert.getValue());

apply_inhomogeneous_kick(staterev, bpertrev, binhomrev);

}
 
//fidelity.close();

//cout << staterev;

cout<< itppextmath::sum_positive_derivatives(listfidel)<< endl;

cout<< real(mean(listcorr))<< endl;
}

if(option2=="correlationandipr"){

itpp::cvec listcorr(steps.getValue());

itpp::cvec init=state;

for(int i=0;i<steps.getValue();i++){

listcorr(i)=pow(abs(dot(conj(init),state)),2);

std::cout << listcorr(i) <<endl;

apply_ising_chain_inhom(state, J.getValue()+Jpert.getValue(), J.getValue()+Jinhompert.getValue()+Jpert.getValue());

apply_inhomogeneous_kick(state, bpert, binhom);

}

cout<< real(mean(listcorr))<< endl;
}

if(option2=="ipr"){

itpp::cvec listcorr(steps.getValue());

itpp::cvec init=state;

for(int i=0;i<steps.getValue();i++){

listcorr(i)=pow(abs(dot(conj(init),state)),2);

//std::cout << listcorr(i) <<endl;

apply_ising_chain_inhom(state, J.getValue()+Jpert.getValue(), J.getValue()+Jinhompert.getValue()+Jpert.getValue());

apply_inhomogeneous_kick(state, bpert, binhom);

}

cout<< real(mean(listcorr))<< endl;
}


}
